#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

#include "headers\vec3.h"

// Cuda Functions:
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if(result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// Rendering Kernel
__global__
void render(vec3* frameBuffer, int max_x, int max_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if((i >= max_x) || (j >= max_y)) {
        return;
    }

    int pixel_index = j * max_x + i;
    frameBuffer[pixel_index] = vec3(float(i) / max_x, float(j) / max_y, 0.25f);
}

int main() {
    // Image Parameters
    const auto aspect_ratio = 1.0f;
    const int image_width = 256;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    const int channels = 3;
    int num_pixels = image_width * image_height;
    size_t frameBuffer_size = num_pixels * sizeof(vec3);

    // Allocate the Frame Buffer and set grid parameters
    vec3 *frameBuffer;
    checkCudaErrors(hipMallocManaged((void **)&frameBuffer, frameBuffer_size));
    int numThreadsX = 8;
    int numThreadsY = 8;
    dim3 blocks(image_width / numThreadsX + 1, image_height / numThreadsY + 1);
    dim3 threads(numThreadsX, numThreadsY);

    // Render the buffer
    clock_t start, stop;
    start = clock();

    render<<<blocks, threads>>>(frameBuffer, image_width, image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "Done! Total render time: " << timer_seconds << "s.\n";

    // Render the image
    unsigned char* image{ new unsigned char[image_width * image_height * channels]{} };
    int idx = 0;
    for(int j = image_height-1; j >= 0; j--) {
        std::cerr << "\rScanlines Remaining: " << j << ' ' << std::flush;
        for(int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            auto r = frameBuffer[pixel_index].x();
            auto g = frameBuffer[pixel_index].y();
            auto b = frameBuffer[pixel_index].z();

            image[idx] = (unsigned char)(255.999f * r);
            image[idx + 1] = (unsigned char)(255.999 * g);
            image[idx + 2] = (unsigned char)(255.999f * b);
            idx += 3;
        }
    }
    
    // Write the pixel array to file, and free the Frame Buffer
    stbi_write_png("images\\colorGradient.png", 
                    image_width, image_height, channels, image, image_width * channels);
    checkCudaErrors(hipFree(frameBuffer));

    return 0;
}
